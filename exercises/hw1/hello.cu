
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello(){

  printf("Hello World from block: %u, thread: %u\n", threadIdx.x, blockIdx.x);
}

int main(){
  hello<<<2,2>>>();
  hipDeviceSynchronize();
}

